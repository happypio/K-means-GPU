#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <iostream>
#include <fstream>
#include <time.h>


#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__constant__ int points_num, clusters_num;

void init_markers(float *points, float *markers, int points_num, int clusters_num) {
    srand(time(NULL));

    int *idxs = (int *)malloc(sizeof(int) * clusters_num);

    for (int i = 0; i < clusters_num; i++) {
        bool unique;
        int point_idx;
        do {
            unique = true;
            point_idx = rand() % points_num;
            for (int j = 0; j < i; j++)
            {
                if (idxs[j] == point_idx)
                    unique = false;
            }
        }
        while(!unique);

        idxs[i] = point_idx;

        markers[2 * i] = points[2 * point_idx];
        markers[2 * i + 1] = points[2 * point_idx + 1];
    }

}

void points_loader(float *points, int points_num)
{
    ifstream inpf("points.txt");
    for (int i = 0; i < points_num; i++)
    {
        inpf >> points[2 * i] >> points[2 * i + 1];
    }
    inpf.close();
}

void points_recorder(float *points, int *assignments, int points_num)
{
    ofstream outf("results_cuda.txt");
    for (int i = 0; i < points_num; i++)
    {
        outf << points[2 * i] << " " << points[2 * i + 1] << " " << assignments[i] << "\n";
    }
    outf.close();
}

__global__ void assign_cluster(float *points, float *markers, int *assignments, float *markers_sums)
{

    extern __shared__ float temp[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid >= points_num) return;

    // load the markers into shared memory
    if (local_tid < clusters_num) {
        temp[2 * local_tid] = markers[2 * local_tid];
        temp[2 * local_tid + 1] = markers[2 * local_tid + 1];
    }

    

    __syncthreads();

    // load the coordinates
    float x = points[2 * tid];
    float y = points[2 * tid  +1];

    float min_distance = (float)INT32_MAX;
    int assigned_cluster = -1;
    
    for (int c = 0; c < clusters_num; c++)
    {
        float distance = (x - temp[2 * c])*(x - temp[2 * c]) 
                        + (y - temp[2 * c + 1])*(y - temp[2 * c + 1]);

        if (distance < min_distance) {
            assigned_cluster = c;
            min_distance = distance;
        }
    }
    
    // update global assignments
    assignments[tid] = assigned_cluster;

    __syncthreads();

    // for each cluster load the coordinates and perform the reduction
    for (int c = 0; c < clusters_num; c++)
    {
        temp[3 * local_tid] = (assigned_cluster == c) ? x : 0;
        temp[3 * local_tid + 1] = (assigned_cluster == c) ? y : 0;
        temp[3 * local_tid + 2] = (assigned_cluster == c) ? 1 : 0;
        __syncthreads();

        for (int i = blockDim.x / 2; i > 0; i /= 2)
        {
            if (local_tid < i)
            {  
                int next_local_tid = local_tid + i;
                temp[3 * local_tid] += temp[3 * next_local_tid];
                temp[3 * local_tid + 1] += temp[3 * next_local_tid + 1];
                temp[3 * local_tid + 2] += temp[3 * next_local_tid + 2];
            }
            __syncthreads();
        }
        
        // now update markers sums for this block
        // we keep data as (x,y,count) per cluster and per block
        if (local_tid == 0)
        {
            markers_sums[3 * clusters_num * blockIdx.x + 3 * c] = temp[3 * local_tid];
            markers_sums[3 * clusters_num * blockIdx.x + 3 * c + 1] = temp[3 * local_tid + 1];
            markers_sums[3 * clusters_num * blockIdx.x + 3 * c + 2] = temp[3 * local_tid + 2];
        }

        // wait for threads, in the next step we will need updated markers_sums
        __syncthreads();
    }
}

__global__ void update_markers(float *markers, float *markers_sums, int num_of_partial_sums)
{
    float sum_x = 0;
    float sum_y = 0;
    float count = 0;

    int index = threadIdx.x;
    // iterate over partial sums starting with idx of thread (#threads == #clusters)
    // (increment by number of clusters)
    for(int i = index; i < num_of_partial_sums; i += clusters_num)
    {
        sum_x += markers_sums[3 * i];
        sum_y += markers_sums[3 * i + 1];
        count += markers_sums[3 * i + 2];
    }

    markers[2 * index] = sum_x / count;
    markers[2 * index + 1] = sum_y / count;
}

int main(int argc, const char **argv)
{
    int h_points_num = atoi(argv[1]), h_clusters_num = atoi(argv[2]), h_max_iter = atoi(argv[3]);
    int num_of_threads = 512;
    // ensure that there is enough blocks
    int num_of_blocks = (h_points_num + num_of_threads - 1) / num_of_threads;
    
    int num_of_partial_sums = num_of_blocks * h_clusters_num;

    // allocate memory on host
    float *points  = (float*)malloc(h_points_num * sizeof(float) * 2);
    int *assignments  = (int*)malloc(h_points_num * sizeof(int));
    float *markers = (float*)malloc(h_clusters_num * sizeof(float)  * 2);

    // initialise card

    findCudaDevice(argc, argv);

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(points_num), &h_points_num, sizeof(h_points_num)));
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(clusters_num), &h_clusters_num, sizeof(h_clusters_num)));

    // allocate device memory
    float *d_points, *d_markers, *d_markers_sum;
    int *d_assignments;

    checkCudaErrors( hipMalloc((void**)&d_points, h_points_num * sizeof(float) * 2) );
    checkCudaErrors( hipMalloc((void**)&d_markers, h_clusters_num * sizeof(float) * 2) );
    checkCudaErrors( hipMalloc((void**)&d_markers_sum, num_of_blocks * h_clusters_num * sizeof(float) * 3) );

    checkCudaErrors( hipMalloc((void**)&d_assignments, h_points_num * sizeof(int) ) );

    // load the points coordinates
    points_loader(points, h_points_num);

    // init markers
    init_markers(points, markers, h_points_num, h_clusters_num);

    // run k means and measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // transfer data onto device
    checkCudaErrors( 
        hipMemcpy(d_points, points, sizeof(float) * h_points_num * 2, hipMemcpyHostToDevice)
    );
    checkCudaErrors( 
        hipMemcpy(d_markers, markers, sizeof(float) * h_clusters_num * 2, hipMemcpyHostToDevice)
    );
    checkCudaErrors( 
        hipMemcpy(d_assignments, assignments, sizeof(int) * h_points_num, hipMemcpyHostToDevice)
    );
    checkCudaErrors(
        hipMemset(d_markers_sum, 0, num_of_blocks * h_clusters_num * sizeof(float) * 3)
    );

    hipEventRecord(start);
    for (int _ = 0; _ < h_max_iter; _++)
    {
        assign_cluster<<<num_of_blocks, num_of_threads, 3 * sizeof(float) * num_of_threads>>>
            (d_points, d_markers, d_assignments, d_markers_sum); 
        hipDeviceSynchronize();
        
        update_markers<<<1, h_clusters_num>>>(d_markers, d_markers_sum, num_of_partial_sums);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);

    hipMemcpy(assignments, d_assignments, sizeof(int) * h_points_num, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    printf("Execution of the K means on GPU: %f milliseconds\n", elapsed_time);
    
    // write to the file
    points_recorder(points, assignments, h_points_num);

    free(points);
    free(markers);
    free(assignments);

    hipFree(d_points);
    hipFree(d_markers);
    hipFree(d_markers_sum);
    hipFree(d_assignments);

    return 0;
}